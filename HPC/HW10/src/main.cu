#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdint.h>
#include <iostream>
#include "../includes/Jpegfile.h"

struct Pixel
{
    uint8_t r;
    uint8_t g;
    uint8_t b;
};

double euclideanDistance(const Pixel *p1, const Pixel *p2)
{
    double sum = 0.0;
    return sqrt(pow(p1->r - p2->r, 2) + pow(p1->g - p2->g, 2) + pow(p1->b - p2->b, 2));
};

__global__ void groupingKernel(UINT width, UINT height, int k, Pixel *colors, Pixel *generators, int *groupColorSum, int *groupCount)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height)
    {
        int index = y * width + x;
        Pixel pixel = colors[index];

        float minDist = INFINITY;
        int minIndex = 0;

        for (int i = 0; i < k; i++)
        {
            Pixel gen = generators[i];
            float dist = sqrtf((pixel.r - gen.r) * (pixel.r - gen.r) + (pixel.g - gen.g) * (pixel.g - gen.g) + (pixel.b - gen.b) * (pixel.b - gen.b));
            if (dist < minDist)
            {
                minDist = dist;
                minIndex = i;
            }
        }

        atomicAdd(&groupColorSum[minIndex * 3 + 0], (int)pixel.r);
        atomicAdd(&groupColorSum[minIndex * 3 + 1], (int)pixel.g);
        atomicAdd(&groupColorSum[minIndex * 3 + 2], (int)pixel.b);
        atomicAdd(&groupCount[minIndex], 1);
    }
}

__global__ void updateGeneratorsKernel(int k, Pixel *generators, int *groupColorSum, int *groupCount)
{
    int i = threadIdx.x;
    if (i < k)
    {
        generators[i].r = (int)groupColorSum[i * 3 + 0] / groupCount[i];
        generators[i].g = (int)groupColorSum[i * 3 + 1] / groupCount[i];
        generators[i].b = (int)groupColorSum[i * 3 + 2] / groupCount[i];
    }
}

__global__ void replaceColorKernel(UINT width, UINT height, int k, Pixel *colors, Pixel *generators, int *groupCount)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height)
    {
        int index = y * width + x;
        Pixel pixel = colors[index];

        float minDist = INFINITY;
        int minIndex = 0;

        for (int i = 0; i < k; i++)
        {
            Pixel gen = generators[i];
            float dist = sqrtf((pixel.r - gen.r) * (pixel.r - gen.r) + (pixel.g - gen.g) * (pixel.g - gen.g) + (pixel.b - gen.b) * (pixel.b - gen.b));
            if (dist < minDist)
            {
                minDist = dist;
                minIndex = i;
            }
        }

        colors[index] = generators[minIndex];
    }
}

int main()
{
    UINT width, height;
    uint8_t *hostDataBuf = JpegFile::JpegFileToRGB("test-large.jpg", &width, &height);

    const int N = height * width; // Total number of pixels

    Pixel *device_colors;

    hipMalloc(&device_colors, width * height * sizeof(Pixel));
    hipMemcpy(device_colors, hostDataBuf, width * height * sizeof(Pixel), hipMemcpyHostToDevice);

    int k;
    std::cout << "Enter the number of clusters: ";
    std::cin >> k;

    auto generators = new Pixel[k];
    printf("Generators: \n");
    for (int i = 0; i < k; i++)
    {
        auto G = generators[i];
        G.r = hostDataBuf[(N * 3 / k) * i];
        G.g = hostDataBuf[(N * 3 / k) * i + 1];
        G.b = hostDataBuf[(N * 3 / k) * i + 2];
        printf("Generator %d: %d %d %d\n", i, G.r, G.g, G.b);
    }

    Pixel *device_generators;
    hipMalloc(&device_generators, k * sizeof(Pixel));
    hipMemcpy(device_generators, generators, k * sizeof(Pixel), hipMemcpyHostToDevice);

    int *device_groupColorSum;
    int *device_groupCount;
    hipMalloc(&device_groupColorSum, k * 3 * sizeof(int));
    hipMalloc(&device_groupCount, k * sizeof(int));
    hipMemset(device_groupColorSum, 0, k * 3 * sizeof(int));
    hipMemset(device_groupCount, 0, k * sizeof(int));

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    for (int iter = 0; iter < 10; iter++)
    {
        // 6. Launch the grouping kernel
        groupingKernel<<<gridSize, blockSize>>>(width, height, k, device_colors, device_generators, device_groupColorSum, device_groupCount);
        hipDeviceSynchronize();

        // 7. Launch the update generators kernel
        updateGeneratorsKernel<<<1, k>>>(k, device_generators, device_groupColorSum, device_groupCount);
        hipDeviceSynchronize();

        // Reset group color sum and count
        hipMemset(device_groupColorSum, 0, k * 3 * sizeof(int));
        hipMemset(device_groupCount, 0, k * sizeof(int));
    }

    replaceColorKernel<<<gridSize, blockSize>>>(width, height, k, device_colors, device_generators, device_groupCount);
    hipDeviceSynchronize();

    hipMemcpy(hostDataBuf, device_colors, N * sizeof(Pixel), hipMemcpyDeviceToHost);

    JpegFile::RGBToJpegFile("output.jpg", hostDataBuf, width, height, 100, false);

    delete[] generators;
    free(hostDataBuf);
    hipFree(device_colors);
    hipFree(device_generators);
    hipFree(device_groupColorSum);
    hipFree(device_groupCount);

    return 0;
}
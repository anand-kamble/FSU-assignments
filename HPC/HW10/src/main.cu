#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // I am not sure if this is needed when using nvcc, but vscode was complaining about not being able to find hip/hip_runtime.h
#include <cmath>
#include <stdint.h>
#include <iostream>
#include "../includes/Jpegfile.h"

// Defining a struct to represent a pixel
struct Pixel
{
    uint8_t r;
    uint8_t g;
    uint8_t b;
};


/**
 * @brief Function to calculate the euclidean distance between two pixels
 * 
 * @param p1 
 * @param p2 
 * @return double 
 */
double euclideanDistance(const Pixel *p1, const Pixel *p2)
{
    double sum = 0.0;
    return sqrt(pow(p1->r - p2->r, 2) + pow(p1->g - p2->g, 2) + pow(p1->b - p2->b, 2));
};


/**
 * @brief Function to calculate the sum of all pixels in a group
 * 
 * @param width 
 * @param height 
 * @param k 
 * @param colors 
 * @param generators 
 * @param groupColorSum 
 * @param groupCount 
 * @return __global__ 
 */
__global__ void groupingKernel(UINT width, UINT height, int k, Pixel *colors, Pixel *generators, int *groupColorSum, int *groupCount)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height)
    {
        int index = y * width + x;
        Pixel pixel = colors[index];

        float minDist = INFINITY;
        int minIndex = 0;

        for (int i = 0; i < k; i++)
        {
            Pixel gen = generators[i];
            float dist = sqrtf((pixel.r - gen.r) * (pixel.r - gen.r) + (pixel.g - gen.g) * (pixel.g - gen.g) + (pixel.b - gen.b) * (pixel.b - gen.b));
            if (dist < minDist)
            {
                minDist = dist;
                minIndex = i;
            }
        }

        atomicAdd(&groupColorSum[minIndex * 3 + 0], (int)pixel.r);
        atomicAdd(&groupColorSum[minIndex * 3 + 1], (int)pixel.g);
        atomicAdd(&groupColorSum[minIndex * 3 + 2], (int)pixel.b);
        atomicAdd(&groupCount[minIndex], 1);
    }
}


/**
 * @brief Function to update the generators
 * 
 * @param k 
 * @param generators 
 * @param groupColorSum 
 * @param groupCount 
 * @return __global__ 
 */
__global__ void updateGeneratorsKernel(int k, Pixel *generators, int *groupColorSum, int *groupCount)
{
    int i = threadIdx.x;
    if (i < k)
    {
        generators[i].r = (int)groupColorSum[i * 3 + 0] / groupCount[i];
        generators[i].g = (int)groupColorSum[i * 3 + 1] / groupCount[i];
        generators[i].b = (int)groupColorSum[i * 3 + 2] / groupCount[i];
    }
}

__global__ void replaceColorKernel(UINT width, UINT height, int k, Pixel *colors, Pixel *generators, int *groupCount)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height)
    {
        int index = y * width + x;
        Pixel pixel = colors[index];

        float minDist = INFINITY;
        int minIndex = 0;

        for (int i = 0; i < k; i++)
        {
            Pixel gen = generators[i];
            float dist = sqrtf((pixel.r - gen.r) * (pixel.r - gen.r) + (pixel.g - gen.g) * (pixel.g - gen.g) + (pixel.b - gen.b) * (pixel.b - gen.b));
            if (dist < minDist)
            {
                minDist = dist;
                minIndex = i;
            }
        }

        colors[index] = generators[minIndex];
    }
}

int main()
{
    UINT width, height;
    uint8_t *hostDataBuf = JpegFile::JpegFileToRGB("test-large.jpg", &width, &height);

    const int N = height * width; // Total number of pixels

    Pixel *device_colors;

    // Record the start time for benchmarking
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipMalloc(&device_colors, width * height * sizeof(Pixel));
    hipMemcpy(device_colors, hostDataBuf, width * height * sizeof(Pixel), hipMemcpyHostToDevice);

    int k;
    std::cout << "Enter the number of clusters: ";
    std::cin >> k;

    auto generators = new Pixel[k];
    printf("Generators: \n");
    for (int i = 0; i < k; i++)
    {
        auto G = generators[i];
        G.r = hostDataBuf[(N * 3 / k) * i];
        G.g = hostDataBuf[(N * 3 / k) * i + 1];
        G.b = hostDataBuf[(N * 3 / k) * i + 2];
        printf("Generator %d: %d %d %d\n", i, G.r, G.g, G.b);
    }

    Pixel *device_generators;
    hipMalloc(&device_generators, k * sizeof(Pixel));
    hipMemcpy(device_generators, generators, k * sizeof(Pixel), hipMemcpyHostToDevice);

    int *device_groupColorSum;
    int *device_groupCount;
    hipMalloc(&device_groupColorSum, k * 3 * sizeof(int));
    hipMalloc(&device_groupCount, k * sizeof(int));
    hipMemset(device_groupColorSum, 0, k * 3 * sizeof(int));
    hipMemset(device_groupCount, 0, k * sizeof(int));

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    for (int iter = 0; iter < 10; iter++)
    {
        // 6. Launch the grouping kernel
        groupingKernel<<<gridSize, blockSize>>>(width, height, k, device_colors, device_generators, device_groupColorSum, device_groupCount);
        hipDeviceSynchronize();

        // 7. Launch the update generators kernel
        updateGeneratorsKernel<<<1, k>>>(k, device_generators, device_groupColorSum, device_groupCount);
        hipDeviceSynchronize();

        // Reset group color sum and count
        hipMemset(device_groupColorSum, 0, k * 3 * sizeof(int));
        hipMemset(device_groupCount, 0, k * sizeof(int));
    }

    replaceColorKernel<<<gridSize, blockSize>>>(width, height, k, device_colors, device_generators, device_groupCount);
    hipDeviceSynchronize();

    hipMemcpy(hostDataBuf, device_colors, N * sizeof(Pixel), hipMemcpyDeviceToHost);

    // Record the end time for benchmarking
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time for CUDA execution: %f ms\n", elapsedTime);

    JpegFile::RGBToJpegFile("output.jpg", hostDataBuf, width, height, 100, false);

    delete[] generators;
    free(hostDataBuf);
    hipFree(device_colors);
    hipFree(device_generators);
    hipFree(device_groupColorSum);
    hipFree(device_groupCount);

    return 0;
}